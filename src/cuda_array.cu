#include "cuda_array.h"

CudaArray::CudaArray(size_t length) : length(length) {
    hipMalloc(&ptr, length * sizeof(scalar_t));
    assert_no_cuda_error();
}

CudaArray::~CudaArray() {
    if (ptr != nullptr) {
        hipFree(ptr);
        assert_no_cuda_error();
    }
}

// Copy data from a CPU buffer into this CUDA array.
void CudaArray::copy_from_range(const scalar_t* src, size_t count) {
    assert(count <= length);
    hipMemcpy(ptr, src, count * sizeof(scalar_t), hipMemcpyDefault);
    assert_no_cuda_error();
}

// Copy the array into a new CPU vector.
std::vector<scalar_t> CudaArray::to_vector() const {
    std::vector<scalar_t> result(length);
    hipMemcpy(result.data(), ptr, length * sizeof(scalar_t), hipMemcpyDefault);
    assert_no_cuda_error();
    return result;
}
